#include "hip/hip_runtime.h"
﻿#include "SandParticle.h"
#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#include "Affine.h"
#include <atomic>
#include <vector>
const int width = 100;
const int height = 10;
const int gridWidth = 800;
const int gridHeight = 800;
const dim3 threadsPerBlock(16, 16);
const int blockSize = 256;
__constant__ float gap = 0.01f;
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define cudaCheck(x) { hipError_t err = x; if (err != hipSuccess) { printf("Cuda error: %d in %s at %s:%d\n", err, #x, __FILE__, __LINE__); assert(0); } }

__global__ void SetGrid(ParticleGrid* grids)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	
	ParticleGrid& grid = grids[index];
	float xPos = -5 + ((index % gridWidth) * gap);
	float yPos = ((index / gridHeight) * gap);

	grid.gridPos.x = xPos;
	grid.gridPos.y = yPos;
	grid.index = index;
	grid.status = Empty;
	grid.predictedMoveInWaterParticleIndex = -1;
	grid.landIndex = -1;
}

__global__ void SetSpawner(ParticleGrid* grid, SpawnerPos* spawnerPos, int startIndex, int width, int i)
{
	int gridIndex = startIndex + threadIdx.x + (i * gridWidth);
	ParticleGrid& currGrid = grid[gridIndex];
	int iIndex = i * width;

	//currGrid.status = FilledWithLand;
	spawnerPos[iIndex + threadIdx.x].pos = currGrid.gridPos;
	spawnerPos[iIndex + threadIdx.x].currGridIndex = currGrid.index;
}

__global__ void SetLand(ParticleGrid* grid, Land* land, int landStartIndex, int landWidth, int i)
{

	int index = landStartIndex + threadIdx.x;
	int ogLandStartIndex = landStartIndex;
	ParticleGrid& currGrid = grid[index];
	int iIndex = i * landWidth;

	currGrid.status = FilledWithLand;
	land[iIndex + threadIdx.x].landPos = currGrid.gridPos;

}

__global__ void CheckGridPos(ParticleSand* particles, ParticleGrid* grids)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	const int startIndex = gridWidth * 500 + 400;

	const int widthVal = index % width;
	const int heightVal = index / width;
	const int heightInGrid = heightVal * gridWidth;
	

	particles[index].currGrid = &grids[startIndex + heightInGrid + widthVal];
	particles[index].pos = grids[startIndex + heightInGrid + widthVal].gridPos;

}

__global__ void SpawnerMove(ParticleGrid* grids, SpawnerPos* spawnerPos, int dir)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	const int currGridIndex = spawnerPos[index].currGridIndex;

	//printf("currGrid : %d", currGridIndex);

	const int destGridIndex = currGridIndex + dir;

	SpawnerPos& spawner = spawnerPos[index];

	spawner.currGridIndex = destGridIndex;
	spawner.pos = grids[destGridIndex].gridPos;
}

__global__ void Spawn(ParticleSand* particle, ParticleGrid* grid, SpawnerPos* spawners, int lastIndex)
{
	int index = lastIndex + threadIdx.x;

	ParticleSand& part = particle[index];
	SpawnerPos& spawner = spawners[threadIdx.x];
	ParticleGrid& gridInfo = grid[spawner.currGridIndex];

	part.pos = spawner.pos;
	part.currGrid = &grid[spawner.currGridIndex];
	gridInfo.status = FilledWithSand;
}

__global__ void SpawnLand(Land* lands, ParticleGrid* grid, SpawnerPos* spawners, int lastIndex)
{
	int index = lastIndex + threadIdx.x + (blockIdx.x * blockDim.x);

	Land& land = lands[index];
	SpawnerPos& spawner = spawners[threadIdx.x];
	ParticleGrid& gridInfo = grid[spawner.currGridIndex];

	land.landPos = spawner.pos;
	land.currGridIndex = spawner.currGridIndex;
	gridInfo.status = FilledWithLand;
}

__global__ void SpawnWater(ParticleWater* waters, ParticleGrid* grid, SpawnerPos* spawners, int lastIndex)
{
	int index = lastIndex + threadIdx.x + (blockIdx.x * blockDim.x);

	ParticleWater& water = waters[index];
	SpawnerPos& spawner = spawners[threadIdx.x];
	ParticleGrid& gridInfo = grid[spawner.currGridIndex];

	water.pos = spawner.pos;
	water.currGridIndex = spawner.currGridIndex;
	gridInfo.status = FilledWithWater;
}

__global__ void MoveWaterFromPredictedPosition(ParticleWater* particles, ParticleGrid* grids, int particleNum)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if(index >= particleNum)
	{
		printf("over");
		return;
	}

	ParticleWater& particle = particles[index];

	if(particle.predictMoveInGridIndex != -1)
	{
		const int predictMoveInGridIndex = particle.predictMoveInGridIndex;
		ParticleGrid& predictedGrid = grids[predictMoveInGridIndex];
		ParticleGrid& currGrid = grids[particle.currGridIndex];
		
		if(predictedGrid.predictedMoveInWaterParticleIndex == index)
		{
			//move
			particle.pos = predictedGrid.gridPos;
			particle.currGridIndex = predictedGrid.index;
			particle.predictMoveInGridIndex = -1;

			predictedGrid.status = FilledWithWater;
			predictedGrid.predictedMoveInWaterParticleIndex = -1;

			currGrid.status = Empty;
		}
		/*else
		{
			const int leftIndex = predictMoveInGridIndex - 1;
			const int rightIndex = predictMoveInGridIndex + 1;
			const int upIndex = predictMoveInGridIndex - gridWidth;
			const int downIndex = predictMoveInGridIndex + gridWidth;

			ParticleGrid& leftGrid = grids[leftIndex];
			ParticleGrid& rightGrid = grids[rightIndex];
			ParticleGrid& downGrid = grids[downIndex];
			ParticleGrid& upGrid = grids[upIndex];
			int newDestGridIndex = -1;

			if(leftGrid.predictedMoveInWaterParticleIndex == -1 && leftGrid.status == Empty)
			{
				newDestGridIndex = leftIndex;
			}
			else if(rightGrid.predictedMoveInWaterParticleIndex == -1 && rightGrid.status == Empty)
			{
				newDestGridIndex = rightIndex;
			}
			else if (downGrid.predictedMoveInWaterParticleIndex == -1 && downGrid.status == Empty)
			{
				newDestGridIndex = downIndex;
			}
			else if (upGrid.predictedMoveInWaterParticleIndex == -1 && upGrid.status == Empty)
			{
				newDestGridIndex = upIndex;
			}

			if(newDestGridIndex != -1)
			{
				ParticleGrid& newDestGrid = grids[newDestGridIndex];
				
				particle.pos = newDestGrid.gridPos;
				particle.currGridIndex = newDestGrid.index;
				particle.predictMoveInGridIndex = -1;

				newDestGrid.status = FilledWithWater;
				newDestGrid.predictedMoveInWaterParticleIndex = -1;

				currGrid.status = Empty;
			}
		}*/

	}

}

__global__ void MoveDownWater(ParticleWater* particles, ParticleGrid* grids, int particleNum)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= particleNum)
	{
		return;
	}

	ParticleWater& particle = particles[index];
	ParticleGrid& currGrid = grids[particle.currGridIndex];

	const int gridIndex = currGrid.index;
	int gridDownIndex = gridIndex - gridWidth;
	int gridLeftDownIndex = gridIndex - (gridWidth - 1);
	int gridRightDownIndex = gridIndex - (gridWidth + 1);
	int gridLeftIndex = gridIndex - 1;
	int gridRightIndex = gridIndex + 1;

	__shared__ ParticleGrid gridsInfo[256 * 2000];
	
	if (gridIndex < gridWidth)
	{
		gridDownIndex = gridIndex;
		gridLeftDownIndex = gridIndex;
		gridRightDownIndex = gridIndex;
		gridLeftIndex = gridIndex;
		gridRightIndex = gridIndex;
	}
	if (gridIndex % gridWidth == 0)
	{
		gridLeftDownIndex = gridIndex;
		gridLeftIndex = gridIndex;
	}
	if (gridIndex % gridWidth == gridWidth - 1)
	{
		gridRightDownIndex = gridIndex;
		gridRightIndex = gridIndex;
	}

	if (gridDownIndex <= 0 || gridDownIndex >= 256 * 2000 ||
		gridLeftDownIndex <= 0 || gridLeftDownIndex >= 256 * 2000 ||
		gridRightDownIndex <= 0 || gridRightDownIndex >= 256 * 2000 || 
		gridLeftIndex <= 0 || gridLeftIndex >= 256 * 2000 ||
		gridRightIndex <= 0 || gridRightIndex >= 256 * 2000)
	{
		return;
	}

	ParticleGrid& downGrid = grids[gridDownIndex];
	ParticleGrid& leftDownGrid = grids[gridLeftDownIndex];
	ParticleGrid& rightDownGrid = grids[gridRightDownIndex];
	ParticleGrid& leftGrid = grids[gridLeftIndex];
	ParticleGrid& rightGrid = grids[gridRightIndex];

	if (downGrid.status == Empty)
	{
		/*particle.pos = downGrid.gridPos;
		particle.currGridIndex = downGrid.index;

		currGrid.status = Empty;
		downGrid.status = FilledWithWater;*/
		downGrid.predictedMoveInWaterParticleIndex = index;
		particle.predictMoveInGridIndex = gridDownIndex;
	}
	else if (leftDownGrid.status == Empty)
	{
		/*particle.pos = leftDownGrid.gridPos;
		particle.currGridIndex = leftDownGrid.index;
		
		currGrid.status = Empty;
		leftDownGrid.status = FilledWithWater;*/
		leftDownGrid.predictedMoveInWaterParticleIndex = index;
		particle.predictMoveInGridIndex = gridLeftDownIndex;
	}
	else if (rightDownGrid.status == Empty)
	{
		//particle.pos = rightDownGrid.gridPos;
		//particle.currGridIndex = rightDownGrid.index;
		//
		//currGrid.status = Empty;
		//rightDownGrid.status = FilledWithWater;
		rightDownGrid.predictedMoveInWaterParticleIndex = index;
		particle.predictMoveInGridIndex = gridRightDownIndex;
	}
	else if (leftGrid.status == Empty)
	{
		
		//particle.pos = grids[gridLeftIndex].gridPos;
		//particle.currGridIndex = grids[gridLeftIndex].index;

		//currGrid.status = Empty;
		//grids[gridLeftIndex].status = FilledWithWater;

		leftGrid.predictedMoveInWaterParticleIndex = index;
		particle.predictMoveInGridIndex = gridLeftIndex;
	}
	else if (rightGrid.status == Empty)
	{
		//particle.pos = grids[gridRightIndex].gridPos;
		//particle.currGridIndex = grids[gridRightIndex].index;

		//currGrid.status = Empty;
		//grids[gridRightIndex].status = FilledWithWater;
		
		rightGrid.predictedMoveInWaterParticleIndex = index;
		particle.predictMoveInGridIndex = gridRightIndex;
	}
	else
	{
		particle.predictMoveInGridIndex = -1;
	}
}

__global__ void MoveDown(ParticleSand* particles, ParticleGrid* grids, int particleNum)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= particleNum)
	{
		return;
	}
	
	ParticleSand& particle = particles[index];
	ParticleGrid& currGrid = *particle.currGrid;

	const int gridIndex = currGrid.index;
	int gridDownIndex = gridIndex - gridWidth;
	int gridLeftDownIndex = gridIndex - (gridWidth - 1);
	int gridRightDownIndex = gridIndex - (gridWidth + 1);

	if (gridIndex < gridWidth)
	{
		gridDownIndex = gridIndex;
		gridLeftDownIndex = gridIndex;
		gridRightDownIndex = gridIndex;
	}
	if (gridIndex % gridWidth == 0)
	{
		gridLeftDownIndex = gridIndex;
	}
	if (gridIndex % gridWidth == gridWidth - 1)
	{
		gridRightDownIndex = gridIndex;
	}

	if (gridDownIndex <= 0 || gridDownIndex >= 256 * 2000 ||
		gridLeftDownIndex <= 0 || gridLeftDownIndex >= 256 * 2000 ||
		gridRightDownIndex <= 0 || gridRightDownIndex >= 256 * 2000)
	{
		return;
	}

	ParticleGrid& downGrid = grids[gridDownIndex];
	ParticleGrid& leftDownGrid = grids[gridLeftDownIndex];
	ParticleGrid& rightDownGrid = grids[gridRightDownIndex];

	if (downGrid.status == Empty)
	{
		particle.pos = downGrid.gridPos;
		particle.currGrid = &downGrid;

		currGrid.status = Empty;
		downGrid.status = FilledWithSand;
	}
	else if (leftDownGrid.status == Empty)
	{
		particle.pos = leftDownGrid.gridPos;
		particle.currGrid = &leftDownGrid;

		currGrid.status = Empty;
		leftDownGrid.status = FilledWithSand;
	}
	else if (rightDownGrid.status == Empty)
	{
		particle.pos = rightDownGrid.gridPos;
		particle.currGrid = &rightDownGrid;

		currGrid.status = Empty;
		rightDownGrid.status = FilledWithSand;
	}
}
__global__ void DeleteLand(Land* lands, ParticleGrid* grids, SpawnerPos* spawners)
{
	int index = threadIdx.x + (blockDim.x * blockIdx.x);

	SpawnerPos& spawnerInfo = spawners[index];
	ParticleGrid& gridInfo = grids[spawnerInfo.currGridIndex];

	if(gridInfo.status == FilledWithLand)
	{
		Land& landInfo = lands[gridInfo.landIndex];

		landInfo.currGridIndex = 0;
		landInfo.landPos = grids[landInfo.currGridIndex].gridPos;
		
		gridInfo.status = Empty;
		gridInfo.landIndex = 0;
	}
}
__global__ void UpdateGrid(ParticleSand* sands, ParticleWater* waters, ParticleGrid* grids)
{
	int index = threadIdx.x + (blockDim.x * blockIdx.x);

	extern __shared__ ParticleGrid gridsInfo[];

	ParticleGrid& grid = grids[index];
	//gridsInfo
	if(grid.status == Empty)
	{
		
	}
}


void DeleteLands(Land* lands, ParticleGrid* grids, SpawnerPos* spawners)
{
	DeleteLand << <1, 256 >> > (lands, grids, spawners);
}

void SimulationUpdate(int particleNum, int gridNum, int waterNum, ParticleSand* particle, ParticleWater* water, ParticleGrid* grid)
{
	int sandGridCount = particleNum / blockSize;
	int waterGridCount = waterNum / blockSize;
	int gridGridCount = gridNum / blockSize;

	
	if(sandGridCount >= 1)
	{
		MoveDown << <sandGridCount, blockSize>> > (particle, grid, particleNum);
		gpuErrchk(hipPeekAtLastError());
	}

	if(waterGridCount >= 1)
	{
		MoveDownWater << <waterGridCount, blockSize >> > (water, grid, waterNum);
		gpuErrchk(hipPeekAtLastError());
		MoveWaterFromPredictedPosition << <waterGridCount, blockSize >> > (water, grid, waterNum);
		gpuErrchk(hipPeekAtLastError());
	}

	//MoveSideWater << <waterGridCount, blockSize >> > (water, grid, waterNum);
	//gpuErrchk(hipPeekAtLastError());
	
}
void AddWatersInSpawnerPos(ParticleWater* waters, ParticleGrid* grid, SpawnerPos* spawners, int lastIndex)
{
	SpawnWater << <1, blockSize >> > (waters, grid, spawners, lastIndex);
}
void AddSandsInContSpawnerPos(ParticleSand* particle, ParticleGrid* grid, Vector2* spawners, int lastIndex)
{
	
}

void AddSandsInSpawnerPos(ParticleSand* particle, ParticleGrid* grid, SpawnerPos* spawners, int lastIndex)
{
	Spawn << <1, blockSize >> > (particle, grid, spawners, lastIndex);
}
void AddLandsInSpawnerPos(Land* lands, ParticleGrid* grid, SpawnerPos* spawners, int lastIndex)
{
	SpawnLand << <1, 64>> > (lands, grid, spawners, lastIndex);
}

void MoveSpawner(ParticleGrid* grid, SpawnerPos* spawners, int dir, int spawnerCount)
{
	SpawnerMove << <1, spawnerCount >> > (grid, spawners, dir);
}



void SetLands(std::vector<int>& landStartRandomIndices, ParticleSand* particle, ParticleGrid* grid, Land* land)
{
	size_t vecSize = landStartRandomIndices.size();
	int landCount = 1;
	int landWidth = 50;
	int landStartIndex = gridWidth * 100 + 200;

	for (size_t i = 0; i < vecSize; ++i)
	{
		SetLand << <landCount, landWidth >> > (grid, land, landStartRandomIndices[i], landWidth, i);
	}

}
__global__ void LoadLand(Land* lands, ParticleGrid* grids, int* indices, int lastIndex)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	
	if (index >= lastIndex)
	{
		printf("over");
		return;
	}
	const int gridIndex = indices[index];

	Land& land = lands[index];
	ParticleGrid& grid = grids[gridIndex];
	land.landPos = grid.gridPos;
	land.currGridIndex = grid.index;
	
	grid.status = FilledWithLand;
	grid.landIndex = index;
}

__global__ void SetContiSpawnerPos(SpawnerPos* poses, ParticleGrid* grids, int startIndex)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);

	SpawnerPos& pos = poses[index];

	const int gridIndex = startIndex + index * 2;

	pos.pos = grids[gridIndex].gridPos;
	pos.currGridIndex = gridIndex;
}

void LoadLands(int* landGridIndices, ParticleGrid* grid, Land* land, int landsNum)
{
	int landBlockCount = landsNum / blockSize;

	if (landBlockCount < 0)
		landBlockCount = 1;

	LoadLand << <landBlockCount, blockSize>> > (land, grid, landGridIndices, landsNum);
	gpuErrchk(hipPeekAtLastError());
}

void Init(int particleNum, int gridNum, int spawnerNum, int contiSpawnerNum, ParticleSand* particle, ParticleGrid* grid, Land* land, SpawnerPos* spawners
, SpawnerPos* firstContiSpawner, SpawnerPos* secondContiSpawner)
{
	int particleGridCount = particleNum / blockSize;
	int gridCount = gridNum / blockSize;
	
	int spawnerStartIndex = gridWidth * 300 + 500;

	SetGrid << <gridCount, blockSize >> > (grid);
	//CheckGridPos << <particleGridCount, blockSize >> > (particle, grid);

	int spawnerWidth = sqrt(spawnerNum);
	int spwanerCount = 1;

	for (int i = 0; i < spawnerWidth; ++i)
	{
		SetSpawner << <spwanerCount, spawnerWidth >> > (grid, spawners, spawnerStartIndex, spawnerWidth, i);
	}
	int contiSpawnerGridCount = contiSpawnerNum / blockSize;

	const int firstContiSpawnerStartIndex = gridWidth * 600 + 200;
	const int secondContiSpawnerStartIndex = firstContiSpawnerStartIndex + 1;
	
	SetContiSpawnerPos << <contiSpawnerGridCount, blockSize >> > (firstContiSpawner, grid, firstContiSpawnerStartIndex);
	SetContiSpawnerPos << <contiSpawnerGridCount, blockSize >> > (secondContiSpawner, grid, secondContiSpawnerStartIndex);
	//gpuErrchk(hipPeekAtLastError());
}